#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "realm.h"
#include <hip/hip_runtime.h>

#ifdef USE_CUBLAS
#include <hipblas.h>
#endif

__global__ void empty_kernel() {}

void gpu_kernel_wrapper(hipStream_t stream)
{
  float *d_A, *d_B, *d_C, *h_A, *h_B, *h_C;
  int N = 100;
  size_t size_matrix = sizeof(float) * N * N;
  h_A = (float *)malloc(size_matrix);
  h_B = (float *)malloc(size_matrix);
  h_C = (float *)malloc(size_matrix);
  for(int i = 0; i < N * N; i++) {
    h_A[i] = 1.0;
    h_B[i] = 1.0;
    h_C[i] = 1.0;
  }
  hipMalloc((void **)&d_A, size_matrix);
  hipMalloc((void **)&d_B, size_matrix);
  hipMalloc((void **)&d_C, size_matrix);
  hipMemcpyAsync(d_A, h_A, size_matrix, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_B, h_B, size_matrix, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_C, h_C, size_matrix, hipMemcpyHostToDevice, stream);
#ifdef USE_CUBLAS
  float alpha = 1.0;
  float beta = 1.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta,
              d_C, N);
#else
  empty_kernel<<<1, 1, 0, stream>>>();
#endif
  hipMemcpyAsync(h_A, d_A, size_matrix, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h_B, d_B, size_matrix, hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(h_C, d_C, size_matrix, hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
#ifdef USE_CUBLAS
  hipblasDestroy(handle);
  for(int i = 0; i < N * N; i++) {
    assert(h_C[i] == N + 1.0);
  }
#endif
  free(h_A);
  free(h_B);
  free(h_C);

  hipEvent_t e1, e2;
  hipEventCreate(&e1);
  hipEventCreate(&e2);
  empty_kernel<<<1, 1, 0, stream>>>();
  empty_kernel<<<1, 1, 0, stream>>>();
  hipEventRecord(e1, stream);
  hipEventRecord(e2, stream);
  hipEventSynchronize(e2);
  hipEventDestroy(e1);
  hipEventDestroy(e2);

  // since stream is a realm stream, so it is OK not to sync it
  empty_kernel<<<1, 1, 0, stream>>>();

#ifdef USE_CUBLAS
  hipblasDestroy(handle);
#endif

  // test ptsz
  // hipStream_t s2;
  // hipStreamCreateWithFlags(&s2, hipStreamDefault);
  // hipStreamSynchronize(s2);
  // hipStreamDestroy(s2);
}